#include "hip/hip_runtime.h"
#include "gpu.h"

__global__ void findRow(char* data, char* index)
{
	int num = (blockIdx.x * blockDim.x + threadIdx.x)*4;
	int id2 = num / 4;
	int id = threadIdx.x * 4;
	extern __shared__ char s[];
	s[id] = data[num];
	s[id+1] = data[num+1];
	s[id+2] = data[num+2];
	s[id+3] = data[num+3];
	if (s[id] == '|' || s[id] == '\n')
	{
		index[id2] = 1;
		return;
	}
	if (s[id +1] == '|' || s[id+1] == '\n')
	{
		index[id2] = 2;
		return;
	}
	if (s[id +2] == '|' || s[id+2] == '\n')
	{
		index[id2] = 3;
		return;
	}
	if (s[id +3] == '|' || s[id+3] == '\n')
	{
		index[id2] = 4;
		return;
	}
	else
		index[id2] = 0;
}

void gpu(char* h_data, char* h_index, long int size, int thread)
{
	const int nStreams = 8;
	hipStream_t streams[nStreams];
	long int dsize = size / nStreams;

	hipFree(0);
	for (int i = 0; i < nStreams; i++)
	{
		hipStreamCreate(&streams[i]);
	}	
	hipError_t err = hipSuccess;
//	char* h_index;
//	h_index = (char*)malloc(sizeof(char) * dsize * nStreams);

	for (int i = 0; i < nStreams; i++)
	{
		char *d_data, *d_index;

		int offset = dsize * i;

		//Alloc data on device
		err = hipHostAlloc((void**)&d_data, dsize * sizeof(char), hipHostMallocDefault);
		//err = hipMalloc((void**)&d_data, len * sizeof(char));
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to malloc data: %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		//alloc index on device
		//err = hipMalloc((void**)&d_index, dsize * sizeof(char));
		err = hipHostAlloc((void**)&d_index, dsize * sizeof(char), hipHostMallocDefault);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to malloc index: %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		//memcopy from host to device
		err = hipMemcpyAsync(d_data, h_data+offset, dsize * sizeof(char), hipMemcpyHostToDevice, streams[i]);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy: %s in stream %d\n", hipGetErrorString(err),i);
			exit(EXIT_FAILURE);
		}

		//kernal execute
		int block = dsize / 4096 + 1;
		findRow << < block, 1024, 4096 * sizeof(char) , streams[i] >> > (d_data, d_index);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to findRow: %s in stream %d\n", hipGetErrorString(err),i);
			exit(EXIT_FAILURE);
		}

		//memcopy from device to host
		err = hipMemcpyAsync(h_index+offset/4, d_index, dsize * sizeof(char) / 4, hipMemcpyDeviceToHost, streams[i]);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to return index: %s in stream %d\n", hipGetErrorString(err),i);
			exit(EXIT_FAILURE);
		}


	}
	for(int i=0;i<nStreams;i++)
	{
		hipStreamDestroy(streams[i]);
	}
}

